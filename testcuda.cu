#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>

__global__ void
zaxpy(const double alpha, const double *A, const double *B, double *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = alpha * A[i] + B[i];
    }
}


int
main(int argc, char **argv)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    long numElements = atoi(argv[1]);
    size_t size = numElements * sizeof(double);
    printf("%ld ", numElements);
    srand48(time(0));

    double *h_A = (double *)malloc(size);
    double *h_B = (double *)malloc(size);
    double *h_C = (double *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = 1.5;
        h_B[i] = 2.3;
    }

    // Allocate the device input vector A
    double *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    double *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    double *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;

    //Warmup
    zaxpy<<<blocksPerGrid, threadsPerBlock>>>(24.0, d_A, d_B, d_C, numElements);

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    zaxpy<<<blocksPerGrid, threadsPerBlock>>>(24.0, d_A, d_B, d_C, numElements);
    hipDeviceSynchronize();
    std::chrono::steady_clock::time_point end= std::chrono::steady_clock::now();

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch zaxpy kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // printf("%.1f %.1f %.1f ", h_C[0], h_C[1000], h_C[numElements-1]);
    std::cout << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() <<std::endl;

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs((24.0f*h_A[i]) + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    // Free device global memory
    err = hipFree(d_A);
    err = hipFree(d_B);
    err = hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}

